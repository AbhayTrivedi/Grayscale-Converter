#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <cassert>

#include "stb_image.h"
#include "stb_image_write.cpp"

struct Pixel {
	unsigned char r, g, b, a;
};

void convertImageToGrayCPU(unsigned char *imageRGBA, int width, int height) {

	for (int y = 0; y < height; y++) {
		for (int x = 0; x < width; x++) {
			Pixel* pixelPtr = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
			unsigned char pixelVal = (unsigned char)(pixelPtr->r * 0.2126f + pixelPtr->g * 0.7152f + pixelPtr->b * 0.0722f);
			pixelPtr->r = pixelVal;
			pixelPtr->g = pixelVal;
			pixelPtr->b = pixelVal;
			pixelPtr->a = 255;
		}
	}

}

__global__ void convertImageToGrayGPU(unsigned char *imageRGBA) {

	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = (y * blockDim.x * gridDim.x + x) * 4;

	Pixel* pixelPtr = (Pixel*)&imageRGBA[idx];
	unsigned char pixelVal = (unsigned char)(pixelPtr->r * 0.2126f + pixelPtr->g * 0.7152f + pixelPtr->b * 0.0722f);
	pixelPtr->r = pixelVal;
	pixelPtr->g = pixelVal;
	pixelPtr->b = pixelVal;
	pixelPtr->a = 255;

}


int main(int argc, char** argv) {
	// Cheking Arguments Count
	if (argc < 2) {
		std::cout << "Usage: 02_ImageToGrayScale <filename>" << std::endl;
		return -1;
	}

	// Opening Image 
	int width, height, compCount;

	std::cout << "Loading File..." << std::endl;
	unsigned char* imageData = stbi_load(argv[1], &width, &height, &compCount, 4);
	
	if (!imageData) {
		std::cout << "Falied to open \"" << argv[1] << "\"" << std::endl;
		return -1;
	}
	
	// Validating image sizes
	if (width % 32 || height % 32) {
		// memory leak
		std::cout << "Wight or Height not divisible by 32." << std::endl;
		return -1;
	}

	// -- Doing Processing using CPU
	//std::cout << "Processing..." << std::endl;
	//convertImageToGrayCPU(imageData, width, height);
	

	// Copy data to GPU
	std::cout << "Coping data to GPU..." << std::endl;
	unsigned char* imageDataGPU = nullptr;
	assert(hipMalloc(&imageDataGPU, width * height * 4) == hipSuccess);
	assert(hipMemcpy(imageDataGPU, imageData, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);
	
	// -- Doing Processing using GPU
	std::cout << "Running CUDA Kernel..." << std::endl;
	dim3 blockSize(32, 32);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);

	convertImageToGrayGPU<<<gridSize, blockSize>>>(imageDataGPU);


	// Copy data from GPU
	std::cout << "Coping data from GPU..." << std::endl;
	assert(hipMemcpy(imageData, imageDataGPU, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	
	// Output FileName
	std::string fileOutName = argv[1];
	fileOutName = fileOutName.substr(0, fileOutName.find_last_of('.')) + "_gray.png";

	
	// Write Image Back to Disk
	std::cout << "Writing PNG file to disk..." << std::endl;
	stbi_write_png(fileOutName.c_str(), width, height, 4, imageData, 4 * width);
	
	
	// Closing Imgae
	stbi_image_free(imageData);
	hipFree(imageDataGPU);

	return 0;
}
